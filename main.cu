#include "hip/hip_runtime.h"
#include <iostream>

#include "Integer.cuh"

template <typename T>
__global__ void isPrime(T const* nData, size_t nDataSize, unsigned int const* p, size_t primesSize, int* isPrime, size_t reps = 25)
{
    Integer<T> const n(nData, nData + nDataSize);

    *isPrime = n.isPrime(p, primesSize, reps);
}

int main()
{
    unsigned int* p(nullptr);
    auto r{hipMalloc(&p, sizeof(unsigned int) * primes.size())};
    assert(r == hipSuccess);
    assert(p);
    r = hipMemcpy(p, primes.data(), sizeof(unsigned int) * primes.size(), hipMemcpyHostToDevice);
    assert(r == hipSuccess);

    int* prime(nullptr);
    r = hipMalloc(&prime, sizeof(int));
    assert(r == hipSuccess);
    assert(prime);

    r = hipDeviceSetLimit(hipLimitStackSize, 256 * 256);
    assert(r == hipSuccess);

    r = hipDeviceSetLimit(hipLimitMallocHeapSize, 256 * 256);
    assert(r == hipSuccess);
    
    using T = uintmax_t;
    
    {
        std::cout << "Block #1" << std::endl;
        hipMemset(prime, -1, sizeof(int));

        auto const n(23 * 29_z);

        auto t{std::chrono::steady_clock::now()};
        std::cout << n.isPrime() << std::endl;
        std::cout << std::chrono::duration_cast<std::chrono::milliseconds>(std::chrono::steady_clock::now() - t).count() << " ms" << std::endl;

        T* nData(nullptr);    
        r = hipMalloc(&nData, sizeof(T) * n.bits().size());
        assert(r == hipSuccess);
        assert(nData);
        r = hipMemcpy(nData, n.bits().data(), sizeof(T) * n.bits().size(), hipMemcpyHostToDevice);
        assert(r == hipSuccess);
        
        t = std::chrono::steady_clock::now();
        isPrime<T><<<1, 1>>>(nData, n.bits().size(), p, primes.size(), prime);

        hipDeviceSynchronize();

        int pr(-1);
        r = hipMemcpy(&pr, prime, sizeof(int), hipMemcpyDeviceToHost);
        assert(r == hipSuccess);

        std::cout << pr << std::endl;
        std::cout << std::chrono::duration_cast<std::chrono::milliseconds>(std::chrono::steady_clock::now() - t).count() << " ms" << std::endl;

        hipFree(nData);
    }

    {
        std::cout << "Block #2" << std::endl;
        hipMemset(prime, -1, sizeof(int));

        auto const n(1299709 * 1299721_z);

        auto t{std::chrono::steady_clock::now()};
        std::cout << n.isPrime() << std::endl;
        std::cout << std::chrono::duration_cast<std::chrono::milliseconds>(std::chrono::steady_clock::now() - t).count() << " ms" << std::endl;

        T* nData(nullptr);    
        r = hipMalloc(&nData, sizeof(T) * n.bits().size());
        assert(r == hipSuccess);
        assert(nData);
        r = hipMemcpy(nData, n.bits().data(), sizeof(T) * n.bits().size(), hipMemcpyHostToDevice);
        assert(r == hipSuccess);
        
        t = std::chrono::steady_clock::now();
        isPrime<T><<<1, 1>>>(nData, n.bits().size(), p, primes.size(), prime);

        hipDeviceSynchronize();

        int pr(-1);
        r = hipMemcpy(&pr, prime, sizeof(int), hipMemcpyDeviceToHost);
        assert(r == hipSuccess);

        std::cout << pr << std::endl;
        std::cout << std::chrono::duration_cast<std::chrono::milliseconds>(std::chrono::steady_clock::now() - t).count() << " ms" << std::endl;

        hipFree(nData);
    }

    {
        std::cout << "Block #3" << std::endl;
        hipMemset(prime, -1, sizeof(int));
    
        auto const n(56062005704198360319209_z);

        auto t{std::chrono::steady_clock::now()};
        std::cout << n.isPrime() << std::endl;
        std::cout << std::chrono::duration_cast<std::chrono::milliseconds>(std::chrono::steady_clock::now() - t).count() << " ms" << std::endl;

        T* nData(nullptr);    
        r = hipMalloc(&nData, sizeof(T) * n.bits().size());
        assert(r == hipSuccess);
        assert(nData);
        r = hipMemcpy(nData, n.bits().data(), sizeof(T) * n.bits().size(), hipMemcpyHostToDevice);
        assert(r == hipSuccess);
        
        t = std::chrono::steady_clock::now();
        isPrime<T><<<1, 1>>>(nData, n.bits().size(), p, primes.size(), prime);

        hipDeviceSynchronize();

        int pr(-1);
        r = hipMemcpy(&pr, prime, sizeof(int), hipMemcpyDeviceToHost);
        assert(r == hipSuccess);

        std::cout << pr << std::endl;
        std::cout << std::chrono::duration_cast<std::chrono::milliseconds>(std::chrono::steady_clock::now() - t).count() << " ms" << std::endl;

        hipFree(nData);
    }

    {
        std::cout << "Block #4" << std::endl;
        hipMemset(prime, -1, sizeof(int));

        auto const n(4113101149215104800030529537915953170486139623539759933135949994882770404074832568499_z);

        T* nData(nullptr);    
        r = hipMalloc(&nData, sizeof(T) * n.bits().size());
        assert(r == hipSuccess);
        assert(nData);
        r = hipMemcpy(nData, n.bits().data(), sizeof(T) * n.bits().size(), hipMemcpyHostToDevice);
        assert(r == hipSuccess);
        
        auto const t{std::chrono::steady_clock::now()};
        isPrime<T><<<1, 1>>>(nData, n.bits().size(), p, primes.size(), prime);

        hipDeviceSynchronize();

        int pr(-1);
        r = hipMemcpy(&pr, prime, sizeof(int), hipMemcpyDeviceToHost);
        assert(r == hipSuccess);

        std::cout << pr << std::endl;
        std::cout << std::chrono::duration_cast<std::chrono::milliseconds>(std::chrono::steady_clock::now() - t).count() << " ms" << std::endl;

        hipFree(nData);
    }

    {
        std::cout << "Block #5" << std::endl;
        hipMemset(prime, -1, sizeof(int));

        Integer64 n;
        n.setPrecision(1024 / 64);
        n.setRandom<std::random_device>();
        n.setPositive();
        if (!(n % 2))
            ++n;

            uint64_t* nData(nullptr);    
        r = hipMalloc(&nData, sizeof(uint64_t) * n.bits().size());
        assert(r == hipSuccess);
        assert(nData);
        r = hipMemcpy(nData, n.bits().data(), sizeof(uint64_t) * n.bits().size(), hipMemcpyHostToDevice);
        assert(r == hipSuccess);
        
        auto const t{std::chrono::steady_clock::now()};
        isPrime<uint64_t><<<1, 1>>>(nData, n.bits().size(), p, primes.size(), prime);

        hipDeviceSynchronize();

        int pr(-1);
        r = hipMemcpy(&pr, prime, sizeof(int), hipMemcpyDeviceToHost);
        assert(r == hipSuccess);

        std::cout << pr << std::endl;
        std::cout << std::chrono::duration_cast<std::chrono::milliseconds>(std::chrono::steady_clock::now() - t).count() << " ms" << std::endl;

        hipFree(nData);
    }

    r = hipFree(p);
    assert(r == hipSuccess);
    r = hipFree(prime);
    assert(r == hipSuccess);

    return 0;
}
