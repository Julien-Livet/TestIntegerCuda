#include "hip/hip_runtime.h"
#include <iostream>

#include "Integer.cuh"

template <typename T>
__global__ void isPrime(T const* nData, size_t nDataSize, unsigned int const* p, size_t primesSize, int* isPrime, size_t reps = 25)
{printf("yo0\n");
    Integer<T> const n(nData, nData + nDataSize);
printf("yo1\n");
    *isPrime = n.isPrime(p, primesSize, reps);printf("yo2\n");
}

int main()
{
    unsigned int* p(nullptr);
    auto r{hipMalloc(&p, sizeof(unsigned int) * primes.size())};
    assert(r == hipSuccess);
    assert(p);
    r = hipMemcpy(p, primes.data(), sizeof(unsigned int) * primes.size(), hipMemcpyHostToDevice);
    assert(r == hipSuccess);

    int* prime(nullptr);
    r = hipMalloc(&prime, sizeof(int));
    assert(r == hipSuccess);
    assert(prime);

    using T = uint64_t;
    
    {
        auto const n(23 * 29_z);

        auto t{std::chrono::steady_clock::now()};
        std::cout << n.isPrime() << std::endl;
        std::cout << std::chrono::duration_cast<std::chrono::milliseconds>(std::chrono::steady_clock::now() - t).count() << " ms" << std::endl;

        T* nData(nullptr);    
        r = hipMalloc(&nData, sizeof(T) * n.bits().size());
        assert(r == hipSuccess);
        assert(nData);
        r = hipMemcpy(nData, n.bits().data(), sizeof(T) * n.bits().size(), hipMemcpyHostToDevice);
        assert(r == hipSuccess);
        
        t = std::chrono::steady_clock::now();std::cout << "hey6" << std::endl;
        isPrime<T><<<1, 1>>>(nData, n.bits().size(), p, primes.size(), prime);
std::cout << "hey7" << std::endl;
        hipDeviceSynchronize();
std::cout << "hey8" << std::endl;
        int pr;
        r = hipMemcpy(&pr, prime, sizeof(int), hipMemcpyDeviceToHost);
        assert(r == hipSuccess);

        std::cout << pr << std::endl;
        std::cout << std::chrono::duration_cast<std::chrono::milliseconds>(std::chrono::steady_clock::now() - t).count() << " ms" << std::endl;

        hipFree(nData);

        return 0;
    }
/*
    {
        auto const n(56062005704198360319209_z);
        
        auto t{std::chrono::steady_clock::now()};
        std::cout << n.isPrime() << std::endl;
        std::cout << std::chrono::duration_cast<std::chrono::milliseconds>(std::chrono::steady_clock::now() - t).count() << " ms" << std::endl;
    }

    {
        auto const n(4113101149215104800030529537915953170486139623539759933135949994882770404074832568499_z);

        auto t{std::chrono::steady_clock::now()};
        std::cout << n.isPrime() << std::endl;
        std::cout << std::chrono::duration_cast<std::chrono::milliseconds>(std::chrono::steady_clock::now() - t).count() << " ms" << std::endl;
    }

    {
        Integer64 n;
        n.setPrecision(1024 / 64);
        n.setRandom<std::random_device>();
        n.setPositive();
        if (!(n % 2))
            ++n;

        auto t{std::chrono::steady_clock::now()};
        std::cout << n.isPrime() << std::endl;
        std::cout << std::chrono::duration_cast<std::chrono::milliseconds>(std::chrono::steady_clock::now() - t).count() << " ms" << std::endl;
    }
*/
    r = hipFree(p);
    assert(r == hipSuccess);
    r = hipFree(prime);
    assert(r == hipSuccess);

    return 0;
}
